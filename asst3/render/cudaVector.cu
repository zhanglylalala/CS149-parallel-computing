#include "cudaVector.h"

template<typename T>
__device__ cudaVector<T>::cudaVector()
{
    len = 0;
    current_length = 2;
    scale = 2;
    hipMalloc(&arr, sizeof(T) * current_length);
}

template<typename T>
__device__ cudaVector<T>::~cudaVector()
{
    hipFree(arr);
}

template<typename T>
__device__ int cudaVector<T>::size()
{
    return len;
}

template<typename T>
__device__ void push_back(T val)
{
    if (len == current_length)
    {
        T* tmp = arr;
        current_length *= scale;
        hipMalloc(&arr, sizeof(T) * current_length);
        for (int i = 0; i < len; i++)
        {
            arr[i] = tmp[i];
        }
        hipFree(tmp);
    }
    arr[len++] = val;
}

template<typename T>
__device__ T& cudaVector<T>::operator[] (int n)
{
    return arr[n];
}